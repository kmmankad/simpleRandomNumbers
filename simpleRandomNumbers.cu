/* 
 * Name: Simple cuRand based random number generator
 * File: simpleRandomNumbers.cu 
 * Description: This file contains a simple CUDA kernel to generate
 *              a matrix of distinct random numbers
 * Author: kmmankad (kmmankad@gmail.com kmankad@ncsu.edu)
 * License: MIT License
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

// Pull in the curand headers
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// We'll use the time as seed
#include <ctime>

// The all-important CUDA error
// checking macros
#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != cudaSuccess ) \
{printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
		__FILE__,__LINE__); exit(-1);} 
#define CUDA_CHECK()  if( (cudaPeekAtLastError()) != cudaSuccess ) \
{printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
		__FILE__,__LINE__-1); exit(-1);} 
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK() 
#endif

// X, Y dimensions of the output matrix
#ifndef NumOfRand_X
#define NumOfRand_X 32
#endif

#ifndef NumOfRand_Y
#define NumOfRand_Y 32
#endif

#define NumOfRand (NumOfRand_X * NumOfRand_Y)

// Block Size
#define NUM_THREADS_X 32
#define NUM_THREADS_Y 32

// CUDA Kernel to initialize the random generator 'states'
__global__ void InitRandGen (int RandSeed, hiprandState_t* RandStates){
	int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_num = thread_x * NUM_THREADS_X + thread_y;
	if (thread_num < NumOfRand) {
		// Initialization is much faster if sequence number and offset
		// are kept at zero, and instead a different seed is used.
		// See - https://devtalk.nvidia.com/default/topic/480586/curand-initialization-time/?offset=4
		hiprand_init(RandSeed+thread_num, /* sequence number */ 0, /* sequence offset */ 0, &RandStates[thread_num]);
	}
}

__global__ void RandGen (int* GPUNums, hiprandState_t* RandStates){
	int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_num = thread_x * NUM_THREADS_X + thread_y;
	if (thread_num < NumOfRand){
		GPUNums[thread_num] = hiprand(&RandStates[thread_num]) % 100;
	}
}


int main(){
	// Allocate memory for the array of
	// random numbers that we want
	int CPUNums[NumOfRand];
	int* GPUNums;

	// Define a pointer for the cuRandStates
	hiprandState_t* RandStates;

	// Allocate the memory for the output nums
	CUDA_CALL(hipMalloc((void**) &GPUNums, sizeof(int) * NumOfRand));

	// Allocate memory for the different curandStates on each core
	CUDA_CALL(hipMalloc((void**) &RandStates, sizeof(hiprandState) * NumOfRand));

	// Launch params
	dim3 BlockSize (NUM_THREADS_X, NUM_THREADS_Y, 1);
	dim3 GridSize((NumOfRand_X/NUM_THREADS_X)+1, (NumOfRand_Y/NUM_THREADS_Y)+1, 1);

	// Launch the Initialization kernel
	InitRandGen<<<GridSize,BlockSize>>>(10, RandStates);
	CUDA_CHECK();
	CUDA_CALL( hipDeviceSynchronize() );

	// Launch the actual generator kernel
	RandGen<<<GridSize, BlockSize>>> (GPUNums, RandStates);
	CUDA_CHECK();
	CUDA_CALL( hipDeviceSynchronize());

	// Get the results back to the host mem
	CUDA_CALL(hipMemcpy(CPUNums, GPUNums, NumOfRand*sizeof(int), hipMemcpyDeviceToHost));

	// Just print some for examination
	for (int i=0; i<40; i++){
		printf ("%0d ", CPUNums[i]);
		if(i%10 == 9)  {
			printf(" \n");
		}
	}    
	return 0;
}

